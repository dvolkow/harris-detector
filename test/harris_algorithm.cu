#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include "../EasyBMP/EasyBMP.h"
#include <cstdint>
#include <cstddef>

#define CORRECT_D ((indexI+i>=0)&&(indexJ+j>=0)&&(indexI+i<height)&&(indexJ+j<width)&&((indexI!=0)||(indexJ!=0)))
#define CORRECT_A ((pictureMeans[i*width+j]<pictureMeans[(i+indexI)*width+j+indexJ])||(pictureMeans[i*width+j]==0))
#define CORRECT_tmp ((tmp[i*width+j]<tmp[(i+indexI)*width+j+indexJ])||(tmp[i*width+j]==0))

//#define SHARED_ENABLE 1
//#define SIZE_BLOCK 1024


__device__ __host__ float meanOfR(int * j_, int * i_, std::uint8_t * picturePixels, int * width_, int * height_, float * threshold_)
{
	float a1 = 0.0f, a2 = 0.0f, a3 = 0.0f, a4 = 0.0f, Ix = 0.0f, Iy = 0.0f;
	float temp;

	//---local var set
	int i = *i_;
	int j = *j_;
	int width = *width_;
	int height = *height_;
	float threshold = *threshold_;
 
	if (i > 0) {
		Ix += 2.0f * picturePixels[i - 1 + j * width];
		if (j > 0) {
			Ix += picturePixels[i - 1 + (j - 1) * width];
			Iy += picturePixels[i - 1 + (j - 1) * width];
		}
		else {
			Ix += picturePixels[i - 1 + (j) * width];
			Iy += picturePixels[i - 1 + (j) * width];
		}
		if (j < height - 1) {
			Ix += picturePixels[i - 1 + (j + 1) * width];
			Iy -= picturePixels[i - 1 + (j + 1) * width];
		}
		else {
			Ix += picturePixels[i - 1 + (j) * width];
			Iy -= picturePixels[i - 1 + (j) * width];
		}
	}
	else {
		Ix += 2.0f * picturePixels[i + (j) * width];
		if (j > 0) {
			Ix += picturePixels[i + (j - 1) * width];
			Iy += picturePixels[i + (j - 1) * width];
		}
		else {
			Ix += picturePixels[i + (j) * width];
			Iy += picturePixels[i + (j) * width];
		}
		if (j < height - 1) {
			Ix += picturePixels[i + (j + 1) * width];
			Iy -= picturePixels[i + (j + 1) * width];
		}
		else {
			Ix += picturePixels[i + (j) * width];
			Iy -= picturePixels[i + (j) * width];
		}
	}
 
	if (j > 0)
		Iy += 2.0f * picturePixels[i + (j - 1) * width];
	else
		Iy += 2.0f * picturePixels[i + (j) * width];
	if (i < width - 1) {
		Ix -= 2.0f * picturePixels[i + 1 + (j) * width];
		if (j > 0) {
			Ix -= picturePixels[i + 1 + (j - 1) * width];
			Iy += picturePixels[i + 1 + (j - 1) * width];
		}
		else {
			Ix -= picturePixels[i + 1 + (j) * width];
			Iy += picturePixels[i + 1 + (j) * width];
		}
  		if (j < height - 1) {
			Ix -= picturePixels[i + 1 + (j + 1) * width];
			Iy -= picturePixels[i + 1 + (j + 1) * width];
		}
		else {
			Ix -= picturePixels[i + 1 + (j) * width];
			Iy -= picturePixels[i + 1 + (j) * width];
		}
	}
	else {
		Ix -= 2.0f * picturePixels[i + (j) * width];
		if (j > 0) {
			Ix -= picturePixels[i + (j - 1) * width];
			Iy += picturePixels[i + (j - 1) * width];
		}
		else {
			Ix -= picturePixels[i + (j) * width];
			Iy += picturePixels[i + (j) * width];
		}
		if (j < height - 1) {
			Ix -= picturePixels[i + (j + 1) * width];
			Iy -= picturePixels[i + (j + 1) * width];
		}
		else {
			Ix -= picturePixels[i + (j) * width];
			Iy -= picturePixels[i + (j) * width];
		}
	}
	
	if (j < height - 1)
		Iy -= 2.0f * picturePixels[i + (j + 1) * width];
	else
		Iy -= 2.0f * picturePixels[i + (j) * width];

	a1 = Ix * Ix;
	a2 = Ix * Iy;
	a3 = Ix * Iy;
	a4 = Iy * Iy;
 
	temp = ((a1 + a4 - a3 - a2) + (0.04f * (a1 + a4) * (a1 + a4)));

	return temp > threshold ? temp : 0;
}

//Calc means
__global__ void fillPictMean(std::uint8_t * picturePixels, int * width_, int * height_, float * threshold_, float * pictureMeans)
{
	//--Calc thread ID & local variables
	int width = * width_;
	int height = * height_;
#ifdef SHARED_ENABLE
	__shared__ extern float tmp[];
#endif
        int i = (threadIdx.x + blockIdx.x * blockDim.x) / width;
        int j = (threadIdx.x + blockIdx.x * blockDim.x) % width;

	if (i < height && j < width)
	{
#ifdef SHARED_ENABLE
		tmp[j + i * width] = meanOfR(&i, &j, picturePixels, width_, height_, threshold_);	
		__syncthreads();
		pictureMeans[j + i * width] = tmp[j + i * width];
#else
		pictureMeans[j + i * width] = meanOfR(&i, &j, picturePixels, width_, height_, threshold_);	
#endif
	}
}


//Harris detector on CUDA. Calculating R, comparing with threshold, finding local maxima
__global__ void kernel(std::uint8_t * picturePixels, int * width_, int * height_, float * pictureMeans)
{
	//--Calc thread ID & local variables
	int width = * width_;
	int height = * height_;
        int i = (threadIdx.x + blockIdx.x * blockDim.x) / width;
        int j = (threadIdx.x + blockIdx.x * blockDim.x) % width;

	if(i < height && j < width) 
	{
#ifdef SHARED_ENABLE 
		//load in shared
		tmp[i * width + j] = pictureMeans[i * width + j];
		__syncthreads();
#endif

		bool localMax = 1;
		for (int indexI = -1; indexI < 2; indexI++)
			for (int indexJ = -1; indexJ < 2; indexJ++)
#ifndef SHARED_ENABLE
				if (CORRECT_D && CORRECT_A)
						localMax = 0;
#else
				if (CORRECT_D) 
				{
					tmp[(i + indexI) * width + j + indexJ] = pictureMeans[(i + indexI) * width + j + indexJ];
					__syncthreads();
					if (CORRECT_tmp)
						localMax = 0;
				}
#endif
		picturePixels[i * width + j] = 1 && localMax;
	}
}


//Funtcion to organize CUDA calls (return time hipEventRecord)
float organizeCUDAcall(std::uint8_t *picturePixels, int *width, int *height, float * threshold)
{
	//CUDA events init
	hipEvent_t startGPUTime;
	hipEvent_t endGPUtime;

	hipEventCreate(&startGPUTime);
	hipEventCreate(&endGPUtime);

	hipEventRecord(startGPUTime, 0);
	//Alloc GPU memory
	const int imageSize = (* width) * (* height);
	dim3 threadCount(*(width));
	dim3 blockSize(*(height));

	std::uint8_t * picturePixelsGPU = NULL;
	int * widthGPU = NULL;
	int * heightGPU = NULL;
	float * thresholdGPU = NULL;
	float * pictureMeansG = NULL;

	hipMalloc(&picturePixelsGPU, imageSize * sizeof(std::uint8_t));
	hipMemcpy(picturePixelsGPU, picturePixels, imageSize * sizeof(std::uint8_t), hipMemcpyHostToDevice);

	hipMalloc(&widthGPU, sizeof(int));
	hipMemcpy(widthGPU, width, sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&heightGPU, sizeof(int));
	hipMemcpy(heightGPU, height, sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&thresholdGPU, sizeof(float));
	hipMemcpy(thresholdGPU, threshold, sizeof(float), hipMemcpyHostToDevice);

	hipMalloc(&pictureMeansG, imageSize * sizeof(float));

	//Call kernel
#ifdef SHARED_ENABLE 
	fillPictMean<<<blockSize, threadCount, imageSize * sizeof(float)>>> (picturePixelsGPU, widthGPU, heightGPU, thresholdGPU, pictureMeansG);
#else
	fillPictMean<<<blockSize, threadCount>>> (picturePixelsGPU, widthGPU, heightGPU, thresholdGPU, pictureMeansG);
#endif
	hipDeviceSynchronize();
#ifdef SHARED_ENABLE 
	kernel<<<blockSize, threadCount, imageSize * sizeof(float)>>> (picturePixelsGPU, widthGPU, heightGPU, pictureMeansG);
#else
	kernel<<<blockSize, threadCount>>> (picturePixelsGPU, widthGPU, heightGPU, pictureMeansG);
#endif
	hipDeviceSynchronize();

	//Copy data from device to host
	hipMemcpy(picturePixels, picturePixelsGPU, imageSize * sizeof(std::uint8_t), hipMemcpyDeviceToHost); 

	//Free memory
	hipFree(thresholdGPU);
	hipFree(heightGPU);
	hipFree(widthGPU);
	hipFree(picturePixelsGPU);
	hipFree(pictureMeansG);

	//Stop -- fix time. Now all GPU activities are stopped
	hipEventRecord(endGPUtime, 0);
	float res = 0;
	hipEventSynchronize(endGPUtime);
	hipEventElapsedTime(&res, startGPUTime, endGPUtime);
	return res;
}

//Harris detector on CPU. Finding local maxima
void harris (std::uint8_t * picturePixels, int width, int height, float threshold)
{
	float * pictureMeans;
	pictureMeans = (float*)malloc(sizeof(float)*width*height);

	for (int i = 0; i < height; i++)
		for (int j = 0; j < width; j++)
			pictureMeans[j + i * width] = meanOfR(&i, &j, picturePixels, &width, &height, &threshold);

	for (int i = 0; i < height; i++)
		for (int j = 0; j < width; j++) {
			bool localMax = 1;
			for (int indexI = -1; indexI < 2; indexI++)
				for (int indexJ = -1; indexJ < 2; indexJ++)
					if (CORRECT_D) {
						if (CORRECT_A)
							localMax = 0;
					}
			if (localMax == 1)
				picturePixels[i * width + j] = 1;
			else
				picturePixels[i * width + j] = 0;
		}
}


//Comparing CPU and GPU results
bool areTheResultsEqual(int height, int width, std::uint8_t * picturePixelsGPU, std::uint8_t * picturePixelsCPU)
{
	for (int i=0; i < height; i++)
		for (int j = 0; j < width; j++)
			if (picturePixelsGPU[i * width + j] != picturePixelsCPU[i * width + j])
				return false;
	return true;
}

//===================================MAIN======================================
int main(int argc, char *argv[]) {

	//--Check args
	if (argc != 3) {
		std::cout << "Usage: " << argv[0] << " <filename>\t"<< "<threshold>" << std::endl;
		return 0;
	}

	char * fileName = argv[1];
	std::uint8_t  * picturePixelsCPU, * picturePixelsGPU;
	float threshold = atof(argv[2]);
	bool equalResults = true;
	BMP AnImage;
	
	//loading grayscale image from BMP24 format (using only red channel)
	AnImage.ReadFromFile(fileName);
	int width = AnImage.TellWidth();
	int height = AnImage.TellHeight();
	int n = width * height;
	picturePixelsCPU = (std::uint8_t *)malloc(sizeof(std::uint8_t )*n);
	picturePixelsGPU = (std::uint8_t *)malloc(sizeof(std::uint8_t )*n);
	for (int i = 0;i < height; i++)
		for (int j = 0;j < width; j++) {
			picturePixelsCPU[i * width + j] = AnImage.GetPixel(j, i).Red;
		}

	memcpy(&picturePixelsGPU[0], &picturePixelsCPU[0], n * sizeof(std::uint8_t ));
	
	//Measure time using CUDA events
	hipEvent_t startCPUTime;
	hipEvent_t endCPUtime;
	hipEventCreate(&startCPUTime);
	hipEventCreate(&endCPUtime);

	hipEventRecord(startCPUTime, 0);
	float cpuTime = 0;
	
	//CPU call
	harris(picturePixelsCPU, width, height, threshold);

	hipEventRecord(endCPUtime, 0);
	hipEventSynchronize(endCPUtime);
	hipEventElapsedTime(&cpuTime, startCPUTime, endCPUtime);


	//Saving the resulting CPU image
	RGBApixel redDot;
	redDot.Red = 255;
	redDot.Blue = 0;
	redDot.Green = 0;
	redDot.Alpha = 0;

	for (int i = 0; i < height; i++)
		for (int j = 0; j < width; j++)
			if (picturePixelsCPU[i * width + j] == 1)
				for (int indexI = -0; indexI < 1; indexI++)
					for (int indexJ = -0; indexJ < 1; indexJ++)
						if ((indexI + i >= 0)&&(indexJ + j >= 0)&&(indexI + i < height)&&(indexJ + j < width))
							AnImage.SetPixel(j, i, redDot);
	AnImage.WriteToFile("out.bmp");


	
	//GPU call
	float gpuTime = organizeCUDAcall(&picturePixelsGPU[0], &width, &height, &threshold);
	
	AnImage.ReadFromFile(fileName);
	//--Save GPU-generated image
	for (int i = 0; i < height; i++)
		for (int j = 0;j < width;j++)
			if (picturePixelsGPU[i * width + j] == 1)
				for (int indexI = -0; indexI < 1; indexI++)
					for (int indexJ = -0; indexJ < 1; indexJ++)
						if ((indexI + i >= 0)&&(indexJ + j>= 0)&&(indexI + i<height)&&(indexJ + j < width))
							AnImage.SetPixel(j, i, redDot);
	AnImage.WriteToFile("out_gpu.bmp");



	//checking the results 
	if (!areTheResultsEqual(height, width, picturePixelsGPU, picturePixelsCPU))
		equalResults = false; 

	std::cout << "Compare result: " << equalResults << std::endl;

	//Print out CPU and GPU time
	std::cout << "CPU time: " << cpuTime << std::endl;
	std::cout << "GPU time: " << gpuTime << std::endl;
	return 0;
}
